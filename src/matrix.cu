#include "hip/hip_runtime.h"
#include "matrix.hpp"

hipDeviceProp_t deviceProp;

Matrix::Matrix(void) {
    this->data = new float[1]();
    this->dim[0] = 0;
    this->dim[1] = 0;
}

Matrix::Matrix(size_t rows, size_t cols) {
    this->data = new float[rows * cols]();
    this->dim[0] = rows;
    this->dim[1] = cols;

    memset(this->data, 0, this->dim[0] * this->dim[1] * sizeof(float));
}

Matrix::Matrix(size_t rows, size_t cols,
               const std::initializer_list<float> &initList) {
    assert(initList.size() == rows * cols);

    this->data = new float[rows * cols]();
    this->dim[0] = rows;
    this->dim[1] = cols;

    memcpy(this->data, initList.begin(), sizeof(float) * rows * cols);
}

Matrix::Matrix(const Matrix &other) : dim{other.dim[0], other.dim[1]} {
    size_t totalElements = dim[0] * dim[1];
    data = new float[totalElements];
    memcpy(data, other.data, totalElements * sizeof(float));
}

Matrix::~Matrix(void) {
    if (this->data != nullptr) {
        delete[] this->data;
        this->data = nullptr;
    }
}

size_t Matrix::get_dim(size_t axis) const {
    assert(axis == 1 || axis == 0);
    return this->dim[axis];
}

float &Matrix::at(size_t row, size_t col) {
    assert(row < this->dim[0]);
    assert(col < this->dim[1]);

    return this->data[row * this->dim[1] + col];
}

float &Matrix::operator()(size_t row, size_t col) {
    return this->data[row * this->dim[1] + col];
}

float Matrix::read_at(size_t row, size_t col) const {
    assert(row < this->dim[0]);
    assert(col < this->dim[1]);

    return this->data[row * this->dim[1] + col];
}

void Matrix::randomize(PRNG &prng, float min, float max) {
    assert(this->data != nullptr);
    assert(this->dim[0] != 0 && this->dim[1] != 0);
    assert(max >= min);

    for (int i = 0; i < this->dim[0] * this->dim[1]; i++) {
        this->data[i] =
            (float)prng.generate() / prng.get_max() * (max - min) + min;
    }
}

void Matrix::fill(float value) {
    for (int i = 0; i < this->dim[0] * this->dim[1]; i++) {
        this->data[i] = value;
    }
}

Matrix Matrix::transpose(void) {
    Matrix transposed(this->dim[1], this->dim[0]);

    for (int i = 0; i < this->dim[0]; i++) {
        for (int j = 0; j < this->dim[1]; j++) {
            transposed(j, i) = (*this)(i, j);
        }
    }

    return transposed;
}

float Matrix::sum(void) {
    float total = 0.0f;
    for (int i = 0; i < this->dim[0]; i++) {
        for (int j = 0; j < this->dim[1]; j++) {
            total += (*this)(i, j);
        }
    }

    return total;
}

std::ostream &operator<<(std::ostream &os, const Matrix &m) {
    assert(m.get_dim(0) != 0 && m.get_dim(1) != 0);

    std::ios::fmtflags old_settings = os.flags();

    os.precision(3);
    os.fill(' ');

    os << "┌ ";
    for (int i = 0; i < m.get_dim(1); i++) {
        os << std::setw(6) << " ";
    }
    os << "┐" << std::endl;

    for (int l = 0; l < m.get_dim(0); l++) {
        os << "│ ";
        for (int c = 0; c < m.get_dim(1); c++) {
            os << std::setw(5) << m.read_at(l, c) << " ";
        }
        os << "│" << std::endl;
    }

    os << "└ ";
    for (int i = 0; i < m.get_dim(1); i++) {
        os << std::setw(6) << " ";
    }
    os << "┘" << std::endl;

    os.flags(old_settings);

    return os;
}

void Matrix::assign(const Matrix &other) {
    this->dim[0] = other.dim[0];
    this->dim[1] = other.dim[1];

    if (this->data != nullptr)
        delete[] this->data;

    this->data = new float[this->dim[0] * this->dim[1]]();

    memcpy(this->data, other.data, this->dim[0] * this->dim[1] * sizeof(float));
}

__global__ void addKernel(float *matrix1, float *matrix2, float *result,
                          int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        result[i * cols + j] = matrix1[i * cols + j] + matrix2[i * cols + j];
    }
}

Matrix Matrix::add(Matrix &other) {
    assert(dim[0] == other.get_dim(0) && dim[1] == other.get_dim(1));

    Matrix result(dim[0], dim[1]);

    if (deviceProp.maxThreadsPerBlock > 0) {
        float *d_matrix1, *d_matrix2, *d_result;

        hipMalloc(&d_matrix1, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_matrix2, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_result, dim[0] * dim[1] * sizeof(float));

        hipMemcpy(d_matrix1, data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, other.data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);

        if (deviceProp.maxThreadsPerBlock <= 0) {
            hipGetDeviceProperties(&deviceProp, 0);
        }

        dim3 blockSize(deviceProp.maxThreadsPerBlock, 1);
        dim3 gridSize((dim[0] + blockSize.x - 1) / blockSize.x,
                      (dim[1] + blockSize.y - 1) / blockSize.y);

        addKernel<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_result,
                                           dim[0], dim[1]);

        hipMemcpy(result.data, d_result, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyDeviceToHost);

        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_result);
    } else {
        for (size_t i = 0; i < dim[0]; ++i) {
            for (size_t j = 0; j < dim[1]; ++j) {
                result(i, j) = other(i, j) + (*this)(i, j);
            }
        }
    }

    return result;
}

__global__ void subKernel(float *matrix1, float *matrix2, float *result,
                          int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        result[i * cols + j] = matrix2[i * cols + j] - matrix1[i * cols + j];
    }
}

Matrix Matrix::sub(Matrix &other) {
    assert(dim[0] == other.get_dim(0) && dim[1] == other.get_dim(1));

    Matrix result(dim[0], dim[1]);

    if (deviceProp.maxThreadsPerBlock > 0) {
        float *d_matrix1, *d_matrix2, *d_result;

        hipMalloc(&d_matrix1, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_matrix2, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_result, dim[0] * dim[1] * sizeof(float));

        hipMemcpy(d_matrix1, data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, other.data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);

        if (deviceProp.maxThreadsPerBlock <= 0) {
            hipGetDeviceProperties(&deviceProp, 0);
        }

        dim3 blockSize(deviceProp.maxThreadsPerBlock, 1);
        dim3 gridSize((dim[0] + blockSize.x - 1) / blockSize.x,
                      (dim[1] + blockSize.y - 1) / blockSize.y);

        subKernel<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_result,
                                           dim[0], dim[1]);

        hipMemcpy(result.data, d_result, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyDeviceToHost);

        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_result);
    } else {
        for (size_t i = 0; i < dim[0]; ++i) {
            for (size_t j = 0; j < dim[1]; ++j) {
                result(i, j) = other(i, j) - (*this)(i, j);
            }
        }
    }

    return result;
}

__global__ void mulKernel(float *matrix1, float *matrix2, float *result,
                          int rows1, int cols1, int cols2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows1 && j < cols2) {
        float sum = 0.0f;
        for (int k = 0; k < cols1; ++k) {
            sum += matrix1[i * cols1 + k] * matrix2[k * cols2 + j];
        }
        result[i * cols2 + j] = sum;
    }
}

Matrix Matrix::mul(Matrix &other) {
    assert(dim[1] == other.get_dim(0));

    Matrix result(dim[0], other.get_dim(1));

    if (deviceProp.maxThreadsPerBlock > 0) {
        float *d_matrix1, *d_matrix2, *d_result;

        hipMalloc(&d_matrix1, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_matrix2,
                   other.get_dim(0) * other.get_dim(1) * sizeof(float));
        hipMalloc(&d_result, dim[0] * other.get_dim(1) * sizeof(float));

        hipMemcpy(d_matrix1, data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, other.data,
                   other.get_dim(0) * other.get_dim(1) * sizeof(float),
                   hipMemcpyHostToDevice);

        if (deviceProp.maxThreadsPerBlock <= 0) {
            hipGetDeviceProperties(&deviceProp, 0);
        }

        dim3 blockSize(deviceProp.maxThreadsPerBlock, 1);
        dim3 gridSize((dim[0] + blockSize.x - 1) / blockSize.x,
                      (other.get_dim(1) + blockSize.y - 1) / blockSize.y);

        mulKernel<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_result,
                                           dim[0], dim[1], other.get_dim(1));

        hipMemcpy(result.data, d_result,
                   dim[0] * other.get_dim(1) * sizeof(float),
                   hipMemcpyDeviceToHost);

        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_result);
    } else {
        for (size_t i = 0; i < dim[0]; ++i) {
            for (size_t j = 0; j < other.get_dim(1); ++j) {
                result(i, j) = 0;
                for (size_t k = 0; k < dim[1]; ++k) {
                    result(i, j) += (*this)(i, k) * other(k, j);
                }
            }
        }
    }

    return result;
}

__global__ void mulScalarKernel(float *matrix, float scalar, float *result,
                                int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        result[i * cols + j] = matrix[i * cols + j] * scalar;
    }
}

Matrix Matrix::mul(float a) {
    if (deviceProp.maxThreadsPerBlock > 0) {
        Matrix result(dim[0], dim[1]);
        float *d_matrix, *d_result;

        hipMalloc(&d_matrix, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_result, dim[0] * dim[1] * sizeof(float));

        hipMemcpy(d_matrix, data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);

        if (deviceProp.maxThreadsPerBlock <= 0) {
            hipGetDeviceProperties(&deviceProp, 0);
        }

        dim3 blockSize(deviceProp.maxThreadsPerBlock, 1);
        dim3 gridSize((dim[0] + blockSize.x - 1) / blockSize.x,
                      (dim[1] + blockSize.y - 1) / blockSize.y);

        mulScalarKernel<<<gridSize, blockSize>>>(d_matrix, a, d_result, dim[0],
                                                 dim[1]);

        hipMemcpy(result.data, d_result, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyDeviceToHost);

        hipFree(d_matrix);
        hipFree(d_result);

        return result;
    } else {
        Matrix result(*this);

        for (size_t i = 0; i < dim[0]; ++i) {
            for (size_t j = 0; j < dim[1]; ++j) {
                result(i, j) *= a;
            }
        }
        return result;
    }
}

__global__ void hadamardKernel(float *matrix1, float *matrix2, float *result,
                               int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        result[i * cols + j] = matrix1[i * cols + j] * matrix2[i * cols + j];
    }
}

Matrix Matrix::hadamard(Matrix &other) {
    assert(dim[0] == other.dim[0] && dim[1] == other.dim[1]);

    Matrix result(dim[0], dim[1]);

    if (deviceProp.maxThreadsPerBlock > 0) {
        float *d_matrix1, *d_matrix2, *d_result;

        hipMalloc(&d_matrix1, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_matrix2, other.dim[0] * other.dim[1] * sizeof(float));
        hipMalloc(&d_result, dim[0] * dim[1] * sizeof(float));

        hipMemcpy(d_matrix1, data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, other.data,
                   other.dim[0] * other.dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);

        if (deviceProp.maxThreadsPerBlock <= 0) {
            hipGetDeviceProperties(&deviceProp, 0);
        }

        dim3 blockSize(deviceProp.maxThreadsPerBlock, 1);
        dim3 gridSize((dim[0] + blockSize.x - 1) / blockSize.x,
                      (dim[1] + blockSize.y - 1) / blockSize.y);

        hadamardKernel<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_result,
                                                dim[0], dim[1]);

        hipMemcpy(result.data, d_result, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyDeviceToHost);

        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_result);
    } else {
        for (size_t i = 0; i < this->dim[0]; i++) {
            for (size_t j = 0; j < this->dim[1]; j++) {
                result(i, j) = this->at(i, j) * other.at(i, j);
            }
        }
    }

    return result;
}

__global__ void applyKernel(float *matrix, float *result, int rows, int cols,
                            float (*f)(float)) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        result[i * cols + j] = f(matrix[i * cols + j]);
    }
}

Matrix Matrix::apply(float (*f)(float)) {
    Matrix result(dim[0], dim[1]);

    if (deviceProp.maxThreadsPerBlock) {
        float *d_matrix, *d_result;

        hipMalloc(&d_matrix, dim[0] * dim[1] * sizeof(float));
        hipMalloc(&d_result, dim[0] * dim[1] * sizeof(float));

        hipMemcpy(d_matrix, data, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyHostToDevice);

        if (deviceProp.maxThreadsPerBlock <= 0) {
            hipGetDeviceProperties(&deviceProp, 0);
        }

        dim3 blockSize(deviceProp.maxThreadsPerBlock, 1);
        dim3 gridSize((dim[0] + blockSize.x - 1) / blockSize.x,
                      (dim[1] + blockSize.y - 1) / blockSize.y);

        applyKernel<<<gridSize, blockSize>>>(d_matrix, d_result, dim[0], dim[1],
                                             f);

        hipMemcpy(result.data, d_result, dim[0] * dim[1] * sizeof(float),
                   hipMemcpyDeviceToHost);

        hipFree(d_matrix);
        hipFree(d_result);
    } else {
        for (size_t i = 0; i < dim[0]; ++i) {
            for (size_t j = 0; j < dim[1]; ++j) {
                result(i, j) = f((*this)(i, j));
            }
        }
    }

    return result;
}

Matrix Matrix::expand(Matrix &other, size_t axis) {
    assert(axis == 0 || axis == 1);

    size_t nrows = 0, ncols = 0;

    if (axis == 0) {
        assert(other.dim[1] == dim[1]);

        nrows = dim[0] + other.dim[0];
        ncols = dim[1];
    } else // axis == 1
    {
        assert(other.dim[0] == dim[0]);

        nrows = dim[0];
        ncols = dim[1] + other.dim[1];
    }

    Matrix result(nrows, ncols);

    for (size_t i = 0; i < dim[0]; ++i) {
        for (size_t j = 0; j < dim[1]; ++j) {
            result(i, j) = (*this)(i, j);
        }
    }

    if (axis == 0) {
        for (size_t i = dim[0]; i < nrows; ++i) {
            for (size_t j = 0; j < ncols; ++j) {
                result(i, j) = other(i - dim[0], j);
            }
        }
    } else // axis == 1
    {
        for (size_t i = 0; i < nrows; ++i) {
            for (size_t j = dim[1]; j < ncols; ++j) {
                result(i, j) = other(i, j - dim[1]);
            }
        }
    }

    return result;
}

Matrix Matrix::extract(size_t row0, size_t row1, size_t col0, size_t col1) {
    assert(row1 > row0 && col1 > col0);

    Matrix result(row1 - row0, col1 - col0);

    for (size_t i = 0; i < row1 - row0; ++i) {
        for (size_t j = 0; j < col1 - col0; ++j) {
            result(i, j) = (*this)(row0 + i, col0 + j);
        }
    }

    return result;
}

Matrix &Matrix::operator=(const Matrix &other) {
    if (this != &other) {
        this->assign(other);
    }
    return (*this);
}

Matrix Matrix::operator+(Matrix &other) { return this->add(other); }

Matrix Matrix::operator-(Matrix &other) { return this->sub(other); }

Matrix Matrix::operator*(Matrix &other) { return this->mul(other); }

Matrix Matrix::operator*(float a) { return this->mul(a); }

Matrix &Matrix::operator+=(Matrix &other) {
    (*this) = this->add(other);
    return (*this);
}

Matrix &Matrix::operator-=(Matrix &other) {
    (*this) = this->sub(other);
    return (*this);
}

Matrix &Matrix::operator*=(Matrix &other) {
    (*this) = this->mul(other);
    return (*this);
}

Matrix &Matrix::operator*=(float a) {
    (*this) = this->mul(a);

    return (*this);
}

void Matrix::save(const char *filename) {
    std::ofstream fp(filename, std::ios::binary | std::ios::out);

    if (fp.is_open()) {
        fp.write(reinterpret_cast<const char *>(&(this->dim[0])),
                 sizeof(size_t));
        fp.write(reinterpret_cast<const char *>(&(this->dim[1])),
                 sizeof(size_t));

        fp.write(reinterpret_cast<const char *>(data),
                 dim[0] * dim[1] * sizeof(float));

        fp.close();
    } else {
        std::cerr << "Unable to open file: " << filename << std::endl;
        abort();
    }
}

void Matrix::load(const char *filename) {
    std::ifstream fp(filename, std::ios::in | std::ios::binary);

    if (fp.is_open()) {
        fp.read(reinterpret_cast<char *>(&(this->dim[1])), sizeof(size_t));
        fp.read(reinterpret_cast<char *>(&(this->dim[0])), sizeof(size_t));

        if (this->data) {
            delete[] this->data;
            this->data = nullptr;
        }
        this->data = new float[this->dim[0] * this->dim[1]]();

        fp.read(reinterpret_cast<char *>(this->data),
                this->dim[0] * this->dim[1] * sizeof(float));

        fp.close();
    } else {
        std::cerr << "Unable to open file: " << filename << std::endl;
        abort();
    }
}

float Matrix::trace(void) {
    assert(this->dim[0] == this->dim[1]);

    float ret = 0.0f;
    for (size_t i = 0; i < this->dim[0]; i++) {
        ret += (*this)(i, i);
    }

    return ret;
}
